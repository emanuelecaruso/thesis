#include "hip/hip_runtime.h"
#include "mapper.cuh"
#include <math.h>
#include "utils.h"
#include <stdlib.h>
#include "defs.h"
#include "cuda_utils.cuh"



__global__ void ComputeWeightedGradientSobelImage_kernel(cv::cuda::PtrStepSz<float> image_in, cv::cuda::PtrStepSz<float> image_out, cv::cuda::PtrStepSz<float> weight_matrix){
  int row = blockIdx.x * blockDim.x + threadIdx.x;
  int col = blockIdx.y * blockDim.y + threadIdx.y;

  int rows = blockDim.x*gridDim.x;
  int cols = blockDim.y*gridDim.y;

  bool down = row>=rows-1;
  bool up = row<=0;
  bool right = col>=cols-1;
  bool left = col<=0;

  int row_u = row+up-1;
  int row_d = row-down+1;
  int col_r = col-right+1;
  int col_l = col+left-1;

  float weight=weight_matrix(row,col);
  // float weight=weight_matrix(row,col)*weight_matrix(row_u,col)*weight_matrix(row_d,col)*weight_matrix(row,col_l)*weight_matrix(row_u,col_l)*weight_matrix(row_d,col_l)*weight_matrix(row,col_r)*weight_matrix(row_u,col_r)*weight_matrix(row_d,col_r);

  float ul = weight_matrix(row_u,col_l)*image_in(row_u,col_l)+(1-weight_matrix(row_u,col_l))*image_in(row_d,col_r);
  float u  = weight_matrix(row_u,col)  *image_in(row_u,col)  +(1-weight_matrix(row_u,col)  )*image_in(row_d,col);
  float ur = weight_matrix(row_u,col_r)*image_in(row_u,col_r)+(1-weight_matrix(row_u,col_r))*image_in(row_u,col_r);
  float l  = weight_matrix(row,col_l)  *image_in(row,col_l)  +(1-weight_matrix(row,col_l)  )*image_in(row,col_r);
  float r  = weight_matrix(row,col_r)  *image_in(row,col_r)  +(1-weight_matrix(row,col_r)  )*image_in(row,col_l);
  float dl = weight_matrix(row_d,col_l)*image_in(row_d,col_l)+(1-weight_matrix(row_d,col_l))*image_in(row_u,col_r);
  float d  = weight_matrix(row_d,col)  *image_in(row_d,col)  +(1-weight_matrix(row_d,col)  )*image_in(row_u,col);
  float dr = weight_matrix(row_d,col_r)*image_in(row_d,col_r)+(1-weight_matrix(row_d,col_r))*image_in(row_u,col_l);

  float value_h = -ul-2*u-ur+dl+2*d+dr;
  float value_v = -ul-2*l-dl+ur+2*r+dr;

  // float value_h = -image_in(row_u,col_l)-2*image_in(row_u,col)-image_in(row_u,col_r)+image_in(row_d,col_l)+2*image_in(row_d,col)+image_in(row_d,col_r);
  // float value_v = -image_in(row_u,col_l)-2*image_in(row,col_l)-image_in(row_d,col_l)+image_in(row_u,col_r)+2*image_in(row,col_r)+image_in(row_d,col_r);

  // float value_h = -image_in(row_u,col)+image_in(row_d,col);
  // float value_v = -image_in(row,col_l)+image_in(row,col_r);

  // float value_h_ = image_in(row,col_l)-2*image_in(row,col)+image_in(row,col_r);
  // float value_v_ = image_in(row_u,col)-2*image_in(row,col)+image_in(row_d,col);

  float value_h_ = l-2*image_in(row,col)+r;
  float value_v_ = u-2*image_in(row,col)+d;

  image_out(row,col)= weight*value_h;
  image_out(row,col+cols)=weight*value_v;
  image_out(row+rows,col)= weight*value_h_;
  image_out(row+rows,col+cols)=weight*value_v_;


}


__global__ void ComputeWeightedDivergenceSobelImage_kernel(cv::cuda::PtrStepSz<float> image_in, cv::cuda::PtrStepSz<float> image_out, cv::cuda::PtrStepSz<float> weight_matrix){
  int row = blockIdx.x * blockDim.x + threadIdx.x;
  int col = blockIdx.y * blockDim.y + threadIdx.y;
  int filter_idx = blockIdx.z * blockDim.z + threadIdx.z;

  int rows = blockDim.x*gridDim.x;
  int cols = blockDim.y*gridDim.y;

  bool down = row>=rows-1;
  bool up = row<=0;
  bool right = col>=cols-1;
  bool left = col<=0;

  int row_u = row+up-1;
  int row_d = row-down+1;
  int col_r = col-right+1;
  int col_l = col+left-1;

  float weight=weight_matrix(row,col);
  // float weight=weight_matrix(row,col)*weight_matrix(row_u,col)*weight_matrix(row_d,col)*weight_matrix(row,col_l)*weight_matrix(row_u,col_l)*weight_matrix(row_d,col_l)*weight_matrix(row,col_r)*weight_matrix(row_u,col_r)*weight_matrix(row_d,col_r);


  // float value_h = (-weight_matrix(row_u,col_l)*image_in(row_u,col_l)-2*weight_matrix(row_u,col)*image_in(row_u,col)-weight_matrix(row_u,col_r)*image_in(row_u,col_r)+weight_matrix(row_d,col_l)*image_in(row_d,col_l)+2*weight_matrix(row_d,col)*image_in(row_d,col)+weight_matrix(row_d,col_r)*image_in(row_d,col_r));
  // float value_v = (-weight_matrix(row_u,col_l+cols)*image_in(row_u,col_l+cols)-2*weight_matrix(row,col_l+cols)*image_in(row,col_l+cols)-weight_matrix(row_d,col_l+cols)*image_in(row_d,col_l+cols)+weight_matrix(row_u,col_r+cols)*image_in(row_u,col_r+cols)+2*weight_matrix(row,col_r+cols)*image_in(row,col_r+cols)+weight_matrix(row_d,col_r+cols)*image_in(row_d,col_r+cols));

  float value_h = (-image_in(row_u,col_l)-2*image_in(row_u,col)-image_in(row_u,col_r)+image_in(row_d,col_l)+2*image_in(row_d,col)+image_in(row_d,col_r));
  float value_v = (-image_in(row_u,col_l+cols)-2*image_in(row,col_l+cols)-image_in(row_d,col_l+cols)+image_in(row_u,col_r+cols)+2*image_in(row,col_r+cols)+image_in(row_d,col_r+cols));

  // float value_h = -image_in(row_u,col)+image_in(row_d,col);
  // float value_v = -image_in(row,col_l+cols)+image_in(row,col_r+cols);

  float value_h_ = image_in(rows+row,col_l+cols)-2*image_in(row+rows,col+cols)+image_in(rows+row,col_r+cols);
  float value_v_ = image_in(rows+row_u,col)-2*image_in(rows+row,col)+image_in(rows+row_d,col);

  // float weight=weight_matrix(row,col);
  // // float weight=weight_matrix(row,col)*weight_matrix(row_u,col)*weight_matrix(row_d,col)*weight_matrix(row,col_l)*weight_matrix(row_u,col_l)*weight_matrix(row_d,col_l)*weight_matrix(row,col_r)*weight_matrix(row_u,col_r)*weight_matrix(row_d,col_r);
  //
  // float ul = weight_matrix(row_u,col_l)*image_in(row_u,col_l)+(1-weight_matrix(row_u,col_l))*image_in(row_d,col_r);
  // float u  = weight_matrix(row_u,col)  *image_in(row_u,col)  +(1-weight_matrix(row_u,col)  )*image_in(row_d,col);
  // float ur = weight_matrix(row_u,col_r)*image_in(row_u,col_r)+(1-weight_matrix(row_u,col_r))*image_in(row_u,col_r);
  // float l  = weight_matrix(row,col_l)  *image_in(row,col_l)  +(1-weight_matrix(row,col_l)  )*image_in(row,col_r);
  // float r  = weight_matrix(row,col_r)  *image_in(row,col_r)  +(1-weight_matrix(row,col_r)  )*image_in(row,col_l);
  // float dl = weight_matrix(row_d,col_l)*image_in(row_d,col_l)+(1-weight_matrix(row_d,col_l))*image_in(row_u,col_r);
  // float d  = weight_matrix(row_d,col)  *image_in(row_d,col)  +(1-weight_matrix(row_d,col)  )*image_in(row_u,col);
  // float dr = weight_matrix(row_d,col_r)*image_in(row_d,col_r)+(1-weight_matrix(row_d,col_r))*image_in(row_u,col_l);
  //
  // float value_h = -ul-2*u-ur+dl+2*d+dr;
  // float value_v = -ul-2*l-dl+ur+2*r+dr;
  //
  // // float value_h = -image_in(row_u,col_l)-2*image_in(row_u,col)-image_in(row_u,col_r)+image_in(row_d,col_l)+2*image_in(row_d,col)+image_in(row_d,col_r);
  // // float value_v = -image_in(row_u,col_l)-2*image_in(row,col_l)-image_in(row_d,col_l)+image_in(row_u,col_r)+2*image_in(row,col_r)+image_in(row_d,col_r);
  //
  // // float value_h = -image_in(row_u,col)+image_in(row_d,col);
  // // float value_v = -image_in(row,col_l)+image_in(row,col_r);
  //
  // // float value_h_ = image_in(row,col_l)-2*image_in(row,col)+image_in(row,col_r);
  // // float value_v_ = image_in(row_u,col)-2*image_in(row,col)+image_in(row_d,col);
  //
  // float value_h_ = l-2*image_in(row,col)+r;
  // float value_v_ = u-2*image_in(row,col)+d;

  float a=1;
  float b=1;
  image_out(row,col)=weight*(a*(value_h+value_v)+b*(-value_h_-value_v_));
  // image_out(row,col)=weight*(value_h+value_v);
  // image_out(row,col)=-weight*(value_h_+value_v_);

  // image_out(row,col)=weight*(value_h-value_h_+value_v-value_v_);

}

__global__ void ComputeWeights_kernel(Camera_gpu* camera_r, cv::cuda::PtrStepSz<float> weight_matrix, float alpha, float beta){

  int row = blockIdx.x * blockDim.x + threadIdx.x;
  int col = blockIdx.y * blockDim.y + threadIdx.y;

  int rows = blockDim.x*gridDim.x;
  int cols = blockDim.y*gridDim.y;

  auto rgb_image= camera_r->image_rgb_;

  bool down = row>=rows-1;
  bool up = row<=0;
  bool right = col>=cols-1;
  bool left = col<=0;

  int row_u = row+up-1;
  int row_d = row-down+1;
  int col_r = col-right+1;
  int col_l = col+left-1;

  float grad_rgbr_x=-rgb_image(row_u,col_l).x-2*rgb_image(row_u,col).x-rgb_image(row_u,col_r).x+rgb_image(row_d,col_l).x+2*rgb_image(row_d,col).x+rgb_image(row_d,col_r).x;
  float grad_rgbg_x=-rgb_image(row_u,col_l).y-2*rgb_image(row_u,col).y-rgb_image(row_u,col_r).y+rgb_image(row_d,col_l).y+2*rgb_image(row_d,col).y+rgb_image(row_d,col_r).y;
  float grad_rgbb_x=-rgb_image(row_u,col_l).z-2*rgb_image(row_u,col).z-rgb_image(row_u,col_r).z+rgb_image(row_d,col_l).z+2*rgb_image(row_d,col).z+rgb_image(row_d,col_r).z;

  float grad_rgbr_y=-rgb_image(row_u,col_l).x-2*rgb_image(row,col_l).x-rgb_image(row_d,col_l).x+rgb_image(row_u,col_r).x+2*rgb_image(row,col_r).x+rgb_image(row_d,col_r).x;
  float grad_rgbg_y=-rgb_image(row_u,col_l).y-2*rgb_image(row,col_l).y-rgb_image(row_d,col_l).y+rgb_image(row_u,col_r).y+2*rgb_image(row,col_r).y+rgb_image(row_d,col_r).y;
  float grad_rgbb_y=-rgb_image(row_u,col_l).z-2*rgb_image(row,col_l).z-rgb_image(row_d,col_l).z+rgb_image(row_u,col_r).z+2*rgb_image(row,col_r).z+rgb_image(row_d,col_r).z;

  // // L1 norm
  // float grad_rgbb_norm=abs(grad_rgbr_x)+abs(grad_rgbg_x)+abs(grad_rgbb_x)+abs(grad_rgbr_y)+abs(grad_rgbg_y)+abs(grad_rgbb_y);

  // L2 norm
  float grad_rgbb_norm=grad_rgbr_x*grad_rgbr_x+grad_rgbg_x*grad_rgbg_x+grad_rgbb_x*grad_rgbb_x+grad_rgbr_y*grad_rgbr_y+grad_rgbg_y*grad_rgbg_y+grad_rgbb_y*grad_rgbb_y;
//
  float weight=1;
  if (grad_rgbb_norm>10000)
    weight=0;
  // float weight=exp(-alpha*pow( grad_rgbb_norm, 10));
  // float weight=exp(-alpha* grad_rgbb_norm);
  // float weight=1;

  weight_matrix(row,col)=weight;
  // printf("%f\n", weight );

}




__global__ void prepareCameraForDtam_kernel(Camera_gpu* camera_r, Camera_gpu* camera_m, cv::cuda::PtrStepSz<float3> query_proj_matrix){
  int row = blockIdx.x * blockDim.x + threadIdx.x;
  int col = blockIdx.y * blockDim.y + threadIdx.y;

  Eigen::Vector2i pixel_coords_r(col,row);

  // query point
  Eigen::Vector3f query_p;
  Eigen::Vector2f uv_r;
  camera_r->pixelCoords2uv(pixel_coords_r, uv_r);
  camera_r->pointAtDepth(uv_r, camera_r->max_depth_, query_p);

  // project query point
  Eigen::Vector2f query_p_projected_on_cam_m;
  float query_depth_on_camera_m;
  bool query_in_front = camera_m->projectPoint(query_p, query_p_projected_on_cam_m, query_depth_on_camera_m);

  float3 val;
  if (!query_in_front)
    val = make_float3( -1,-1,-1 );
  else
    val = make_float3( query_p_projected_on_cam_m.x(), query_p_projected_on_cam_m.y(), query_depth_on_camera_m );

  query_proj_matrix(row,col)=val;

}


void Mapper::prepareCameraForDtam(int index_m){
  Camera_cpu* camera_r = camera_vector_cpu_[index_r_];
  Eigen::Vector3f camera_r_p = camera_r->frame_camera_wrt_world_->translation();
  Camera_cpu* camera_m = camera_vector_cpu_[index_m];

  int cols = camera_r->invdepth_map_->image_.cols;
  int rows = camera_r->invdepth_map_->image_.rows;

  // project camera_r on camera_m
  Eigen::Vector2f cam_r_projected_on_cam_m;
  float cam_r_depth_on_camera_m;
  bool cam_r_in_front = camera_m->projectPoint(camera_r_p, cam_r_projected_on_cam_m, cam_r_depth_on_camera_m);

  Eigen::Isometry3f T = (*(camera_m->frame_world_wrt_camera_))*(*(camera_r->frame_camera_wrt_world_));
  Eigen::Matrix3f r=T.linear();
  Eigen::Vector3f t=T.translation();

  cameraDataForDtam* camera_data_for_dtam_h = new cameraDataForDtam;
  camera_data_for_dtam_h->T_r=r;
  camera_data_for_dtam_h->T_t=t;
  camera_data_for_dtam_h->cam_r_projected_on_cam_m=cam_r_projected_on_cam_m;
  camera_data_for_dtam_h->cam_r_depth_on_camera_m=cam_r_depth_on_camera_m;
  camera_data_for_dtam_h->cam_r_in_front=cam_r_in_front;
  query_proj_matrix_.create(rows,cols,CV_32FC3);
  camera_data_for_dtam_h->query_proj_matrix=query_proj_matrix_;



  // Kernel invocation
  dim3 threadsPerBlock( 8 , 8 , 1);
  dim3 numBlocks( rows/8, cols/8 , 1);
  prepareCameraForDtam_kernel<<<numBlocks,threadsPerBlock>>>( camera_r->camera_gpu_, camera_m->camera_gpu_, camera_data_for_dtam_h->query_proj_matrix);
  printCudaError("Kernel preparing camera for dtam "+camera_m->name_);

  hipMalloc(&camera_data_for_dtam_, sizeof(cameraDataForDtam));
  printCudaError("hipMalloc (dtam constr) "+camera_m->name_);

  hipMemcpy(camera_data_for_dtam_, camera_data_for_dtam_h, sizeof(cameraDataForDtam), hipMemcpyHostToDevice);
  printCudaError("hipMemcpy (dtam constr) "+camera_m->name_);

  delete camera_data_for_dtam_h;

}


__global__ void UpdateCostVolume_kernel(Camera_gpu* camera_r, Camera_gpu* camera_m,
              cv::cuda::PtrStepSz<int2> cost_volume, cameraDataForDtam* camera_data_for_dtam_,
              float* invdepth_r_array, int threshold, bool occl){


  int row = blockIdx.x * blockDim.x + threadIdx.x;
  int col = blockIdx.y * blockDim.y + threadIdx.y;
  int i = blockIdx.z * blockDim.z + threadIdx.z;

  int cols = blockDim.y*gridDim.y;

  // initializations
  Eigen::Vector2f uv_r;
  Eigen::Vector2i pixel_coords_r(col,row);
  camera_r->pixelCoords2uv(pixel_coords_r, uv_r);
  bool stop = false;

  uchar3 clr_r = camera_r->image_rgb_(row,col);
  float depth1_r=camera_r->lens_;
  float depth2_r=camera_r->max_depth_;
  float3 val = camera_data_for_dtam_->query_proj_matrix(row,col);
  if (val.z<0)
    stop = true;
  Eigen::Vector2f uv1_fixed = camera_data_for_dtam_->cam_r_projected_on_cam_m;
  Eigen::Vector2f uv2_fixed;
  uv2_fixed.x()=val.x;
  uv2_fixed.y()=val.y;
  float depth1_m_fixed = camera_data_for_dtam_->cam_r_depth_on_camera_m;
  float depth2_m_fixed = val.z;
  Eigen::Matrix3f r=camera_data_for_dtam_->T_r;
  Eigen::Vector3f t=camera_data_for_dtam_->T_t;
  float f = camera_m->lens_;
  float w=camera_m->width_;
  float h=camera_m->width_/camera_m->aspect_;

  Eigen::Vector2i pixel_current;

  if(!stop){

    float depth_r = 1.0/invdepth_r_array[i];

    float depth_m = depth_r*r(2,2)-t(2)-((depth_r*r(2,0)*(2*uv_r.x()-w))/(2*f))-((depth_r*r(2,1)*(-2*uv_r.y()+h))/(2*f));
    float ratio_invdepth_m = ((1.0/depth_m)-(1.0/depth1_m_fixed))/((1.0/depth2_m_fixed)-(1.0/depth1_m_fixed));

    Eigen::Vector2f uv_current;
    uv_current.x()=uv1_fixed.x()+ratio_invdepth_m*(uv2_fixed.x()-uv1_fixed.x()) ;
    uv_current.y()=uv1_fixed.y()+ratio_invdepth_m*(uv2_fixed.y()-uv1_fixed.y()) ;

    camera_m->uv2pixelCoords( uv_current, pixel_current);

    if(pixel_current.x()<0 || pixel_current.y()<0 || pixel_current.x()>=(camera_m->resolution_) || pixel_current.y()>=(int)((float)camera_m->resolution_/(float)camera_m->aspect_) )
      stop=true;
  }

  int col_ = cols*i+col;

  if (!stop){

    uchar3 clr_current = camera_m->image_rgb_(pixel_current.y(),pixel_current.x());

    // int cost_current=((clr_r.x-clr_current.x)*(clr_r.x-clr_current.x)+(clr_r.y-clr_current.y)*(clr_r.y-clr_current.y)+(clr_r.z-clr_current.z)*(clr_r.z-clr_current.z));
    int cost_current=(abs(clr_r.x-clr_current.x)+abs(clr_r.y-clr_current.y)+abs(clr_r.z-clr_current.z));

    if(occl){
      cost_current=min(cost_current, 20);
      // cost_current=max(cost_current, 3);
    }

    if (cost_current<20 || occl)
    {
        // cost_current=max(cost_current, 3);
        // cost_current=min(cost_current, 50);

        int2 cost_volume_val = cost_volume(row,col_);

        cost_volume_val.x = (cost_volume_val.x*cost_volume_val.y+cost_current)/(cost_volume_val.y+1);

        cost_volume_val.y++;

        cost_volume(row,col_) = cost_volume_val;
    }

  }

}



__global__ void StudyCostVolumeMin_kernel(Camera_gpu* camera_r, Camera_gpu* camera_m,
              cv::cuda::PtrStepSz<int2> cost_volume, cameraDataForDtam* camera_data_for_dtam_, float* invdepth_r_array,
              int row, int col, cv::cuda::PtrStepSz<float> depth_groundtruth, cv::cuda::PtrStepSz<float> a){


  int i = blockIdx.z * blockDim.z + threadIdx.z;

  int cols = camera_r->resolution_;

  // initializations
  Eigen::Vector2f uv_r;
  Eigen::Vector2i pixel_coords_r(col,row);
  camera_r->pixelCoords2uv(pixel_coords_r, uv_r);

  bool stop = false;

  uchar3 clr_r = camera_r->image_rgb_(row,col);
  float depth1_r=camera_r->lens_;
  float depth2_r=camera_r->max_depth_;
  float3 val = camera_data_for_dtam_->query_proj_matrix(row,col);
  if (val.z<0)
    stop = true;
  Eigen::Vector2f uv1_fixed = camera_data_for_dtam_->cam_r_projected_on_cam_m;
  Eigen::Vector2f uv2_fixed;
  uv2_fixed.x()=val.x;
  uv2_fixed.y()=val.y;
  float depth1_m_fixed = camera_data_for_dtam_->cam_r_depth_on_camera_m;
  float depth2_m_fixed = val.z;
  Eigen::Matrix3f r=camera_data_for_dtam_->T_r;
  Eigen::Vector3f t=camera_data_for_dtam_->T_t;
  float f = camera_m->lens_;
  float w=camera_m->width_;
  float h=camera_m->width_/camera_m->aspect_;

  Eigen::Vector2i pixel_current;

  if(!stop){

    float depth_r = 1.0/invdepth_r_array[i];

    // Eigen::Vector3f p;
    // float depth_m;
    // camera_r->pointAtDepth(uv_r, depth_r, p);
    // Eigen::Vector2f uv_current;
    // bool query_in_front = camera_m->projectPoint(p, uv_current, depth_m);

    float depth_m = depth_r*r(2,2)-t(2)-((depth_r*r(2,0)*(2*uv_r.x()-w))/(2*f))-((depth_r*r(2,1)*(-2*uv_r.y()+h))/(2*f));
    float ratio_invdepth_m = ((1.0/depth_m)-(1.0/depth1_m_fixed))/((1.0/depth2_m_fixed)-(1.0/depth1_m_fixed));
    // float ratio_invdepth_m = ((1.0/depth_m)-(1.0/depth1_m_fixed))/((1.0/depth2_m_fixed)-(1.0/depth1_m_fixed));

    // printf("%f\n", depth_m );

    Eigen::Vector2f uv_current;
    uv_current.x()=uv1_fixed.x()+ratio_invdepth_m*(uv2_fixed.x()-uv1_fixed.x()) ;
    uv_current.y()=uv1_fixed.y()+ratio_invdepth_m*(uv2_fixed.y()-uv1_fixed.y()) ;

    camera_m->uv2pixelCoords( uv_current, pixel_current);

    if(pixel_current.x()<0 || pixel_current.y()<0 || pixel_current.x()>=(camera_m->resolution_) || pixel_current.y()>=(int)((float)camera_m->resolution_/(float)camera_m->aspect_) )
      stop=true;
  }

  int col_ = cols*i+col;

  int cost_current;
  uchar3 clr_current;

  if (!stop){

    clr_current = camera_m->image_rgb_(pixel_current.y(),pixel_current.x());

    // int cost_current=((clr_r.x-clr_current.x)*(clr_r.x-clr_current.x)+(clr_r.y-clr_current.y)*(clr_r.y-clr_current.y)+(clr_r.z-clr_current.z)*(clr_r.z-clr_current.z));
    cost_current=(abs(clr_r.x-clr_current.x)+abs(clr_r.y-clr_current.y)+abs(clr_r.z-clr_current.z));

    int val =255-(cost_current/3);
    uchar3 magenta = make_uchar3(255,0,255);
    uchar3 magenta_ = make_uchar3(val,0,val);

    camera_m->image_rgb_(pixel_current.y(),pixel_current.x())=magenta;

  }

  __syncthreads();

  __shared__ int cost_array[NUM_INTERPOLATIONS];
  __shared__ int indx_array[NUM_INTERPOLATIONS];

  cost_array[i]=cost_volume(row,col_).x;
  indx_array[i]=i;

  // REDUCTION
  // Iterate of log base 2 the block dimension
  for (int s = 1; s < NUM_INTERPOLATIONS; s *= 2) {
    // Reduce the threads performing work by half previous the previous
    // iteration each cycle
    if (i % (2 * s) == 0) {
      int min_cost = min(cost_array[i + s], cost_array[i]);

      if (cost_array[i] > min_cost){
        indx_array[i] = indx_array[i+s];
        cost_array[i] = min_cost ;
      }
    }
    __syncthreads();
  }
  __syncthreads();
  // printf("cost current at i=%i is: %i\n",i, cost_current);

  if (i == indx_array[0]) {
  // if (i == 57) {
  // if (i == 1) {
    uchar3 blue = make_uchar3(255,0,0);
    camera_m->image_rgb_(pixel_current.y(),pixel_current.x())=blue;

    // printf("camera_r->max_depth_: %f\n", camera_r->max_depth_);
    // printf("depth1_m_fixed: %f\n", depth1_m_fixed);
    // printf("depth2_m_fixed: %f\n", depth2_m_fixed);
    printf("predicted depth from costvol: %f\n", 1.0/invdepth_r_array[indx_array[0]]);
    printf("predicted depth from a: %f\n", 1.0/(a(row,col)*invdepth_r_array[0]));
    printf("grondtruth depth: %f\n", (1.0/(depth_groundtruth(row,col)*2)));
    // printf("grondtruth val: %f\n", depth_groundtruth(row,col));
    printf("clr_r is: %i,%i,%i\n", clr_r.x ,clr_r.y ,clr_r.z);
    printf("clr_current is: %i,%i,%i\n", clr_current.x ,clr_current.y ,clr_current.z);
    // printf("stop flag is: %i\n", stop);
    printf("cost current is: %i\n", cost_current);
    printf("cost current TOT is: %i\n", cost_volume(row,col_).x);
    printf("n projections: %i\n", cost_volume(row,col_).y);
    // printf("cost current is: %i\n", cost_current);
    printf("min cost is: %i\n", cost_array[0]);
    // printf("coords: %i %i\n", pixel_current.y(), pixel_current.x());
    printf("idx: %i\n", i);
    printf("\n");
  }

}


__global__ void ComputeCostVolumeMin_kernel( cv::cuda::PtrStepSz<float> d, cv::cuda::PtrStepSz<float> a, cv::cuda::PtrStepSz<int2> cost_volume, float* invdepth_r_array){

  int row = blockIdx.x * blockDim.x + threadIdx.x;
  int col = blockIdx.y * blockDim.y + threadIdx.y;
  int i = blockIdx.z * blockDim.z + threadIdx.z;

  int cols = blockDim.y*gridDim.y;
  int col_ = cols*i+col;


  __shared__ int cost_array[4][4][NUM_INTERPOLATIONS];
  __shared__ int indx_array[4][4][NUM_INTERPOLATIONS];

  cost_array[threadIdx.x][threadIdx.y][i]=cost_volume(row,col_).x;
  indx_array[threadIdx.x][threadIdx.y][i]=i;

  // REDUCTION
  // Iterate of log base 2 the block dimension
	for (int s = 1; s < NUM_INTERPOLATIONS; s *= 2) {
		// Reduce the threads performing work by half previous the previous
		// iteration each cycle
		if (i % (2 * s) == 0) {
      int min_cost = min(cost_array[threadIdx.x][threadIdx.y][i + s], cost_array[threadIdx.x][threadIdx.y][i]);
      if (cost_array[threadIdx.x][threadIdx.y][i] > min_cost){
        indx_array[threadIdx.x][threadIdx.y][i] = indx_array[threadIdx.x][threadIdx.y][i+s];
        cost_array[threadIdx.x][threadIdx.y][i] = min_cost ;
      }
		}
		__syncthreads();
	}
  if (i == indx_array[threadIdx.x][threadIdx.y][0]) {

    // printf("%i\n", cost_array[threadIdx.x][threadIdx.y][0] );

    float val =(float)invdepth_r_array[i]/invdepth_r_array[0];

    d(row,col)=val;
    a(row,col)=val;
    if (indx_array[threadIdx.x][threadIdx.y][0]==0){
      d(row,col)=1.0;
      a(row,col)=1.0;
    }
	}

}


__global__ void Image2Vector_kernel(cv::cuda::PtrStepSz<float> image, float* vector){

  int row = blockIdx.x * blockDim.x + threadIdx.x;
  int col = blockIdx.y * blockDim.y + threadIdx.y;

  int rows = blockDim.x*gridDim.x;
  int cols = blockDim.y*gridDim.y;

  int index = row+col*rows;
  vector[index]=image(row,col);

}

__global__ void UpdateDepthmap_kernel(Camera_gpu* camera, cv::cuda::PtrStepSz<int2> cost_volume, cv::cuda::PtrStepSz<float> weight_matrix, cv::cuda::PtrStepSz<float> a, cv::cuda::PtrStepSz<float> d, cv::cuda::PtrStepSz<float> gradient_q, int switch_idx, float switch_depth, float depth1_r, float depth2_r){
  int row = blockIdx.x * blockDim.x + threadIdx.x;
  int col = blockIdx.y * blockDim.y + threadIdx.y;
  int rows = blockDim.x*gridDim.x;
  int cols = blockDim.y*gridDim.y;
  int index = row+col*rows;

  camera->invdepth_map_(row,col)=0;
  // points_added(row,col)=0;
  // if(abs(gradient_q(row,col))<0.01){
  if(abs(gradient_q(row,col))<0.1 && weight_matrix(row,col)>0.8){
  // if(true){

    float invdepth=d(row,col);
    // float invdepth=a(row,col);
    float depth=depth1_r/invdepth;
    int i;
    if(depth<switch_depth){
      i= (int)roundf( switch_idx*((depth -depth1_r)/(switch_depth-depth1_r)));
      // if(i<0)
        // printf("i:%i, d: %f\n",i, invdepth);
    }
    else{
      i=switch_idx+(int)roundf(((NUM_INTERPOLATIONS-switch_idx-1)*((1.0/depth)-(1.0/switch_depth)))/((1.0/depth2_r)-(1.0/switch_depth)));
    }

    if (i>=0){
      int col_ = cols*i+col;
      int cost = cost_volume(row,col_).x;
      int nproj = cost_volume(row,col_).y;

      // if(nproj>2 && cost<10){
      if(nproj>2){
      // if(true){
        camera->invdepth_map_(row,col)=d(row,col);
        // camera->invdepth_map_(row,col)=a(row,col);

      }
    }
  }

}

__global__ void PopulateState_kernel(Camera_gpu* camera){
  int row = blockIdx.x * blockDim.x + threadIdx.x;
  int col = blockIdx.y * blockDim.y + threadIdx.y;
  int rows = blockDim.x*gridDim.x;
  int cols = blockDim.y*gridDim.y;
  int index = row+col*rows;

  Eigen::Vector2f uv_r;
  Eigen::Vector2i pixel_coords_r(col,row);
  camera->pixelCoords2uv(pixel_coords_r, uv_r);
  float depth=1.0/(camera->invdepth_map_(row,col)*(1.0/camera->min_depth_));
  Eigen::Vector3f p;


  bool valid;
  if(camera->invdepth_map_(row,col)!=0){
    valid=true;
    camera->pointAtDepth(uv_r, depth, p);
  }
  else{
    valid=false;
    p={0,0,0};
  }
  uchar3 clr = camera->image_rgb_(row,col);

  Cp_gpu cp = {p, {clr.x,clr.y,clr.z}, valid};

  camera->cp_array_[index]=cp;

  // if (index<=100)
  // printf("%f %i %i\n", camera->cp_array_[index].point.x(), valid, index );

}

void Mapper::Initialize(){

  threshold_=50;
  // theta_=0.07;
  // theta_=0.1;
  theta_=0.3;
  // theta_=1000;
  // theta_=0.5;
  theta_switch_=0057;
  theta_switch_=0;
  theta_end_=0.00001;
  eps_=0.01;
  // eps_=0.001;
  // alpha_=0.0000000003;
  alpha_=0.000000000000000000000001;
  // alpha_=0;
  // alpha_=0.00002;
  // alpha_=0.00003;
  // alpha_=0.0005;
  beta1_=0.0;
  // beta1_=0.0005;
  beta2_=0.01;
  // beta2_=0.0001;
  // lambda_=0.1;
  lambda_=0.01;
  // lambda_=0.0005;
  sigma_q0_=0.1;
  sigma_d0_=0.5;

  // r1_=1;
  r1_=0.97;
  // r1_=0.99;
  // r2_=0.7;
  r2_=0.97;


  n_ = 0;
  sigma_q_=sigma_q0_;
  sigma_d_=sigma_d0_;

  int cols = camera_vector_cpu_[index_r_]->invdepth_map_->image_.cols;
  int rows = camera_vector_cpu_[index_r_]->invdepth_map_->image_.rows;

  a.create(rows,cols,CV_32FC1);
  d.create(rows,cols,CV_32FC1);
  q.create(rows*2,cols*2,CV_32FC1);
  gradient_d.create(rows*2,cols*2,CV_32FC1);
  gradient_q.create(rows,cols,CV_32FC1);

  // depth_groundtruth_.create(rows,cols,CV_32FC1);
  depth_groundtruth_ = camera_vector_cpu_[index_r_]->invdepth_map_gpu_.clone();
  camera_vector_cpu_[index_r_]->invdepth_map_gpu_.setTo(0);
  camera_vector_cpu_[index_r_]->invdepth_map_->setAllPixels(0);

  cost_volume_.create(rows,cols*NUM_INTERPOLATIONS,CV_32SC2);
  cost_volume_.setTo(cv::Scalar(INT_MAX,0));

  weight_matrix_.create(rows,cols,CV_32FC1);

  Mapper::ComputeWeights();

}

void Mapper::UpdateCostVolume(int index_m, bool occl){
  double t_s=getTime();

  Camera_cpu* camera_r_cpu = camera_vector_cpu_[index_r_];
  Camera_gpu* camera_r_gpu = camera_r_cpu->camera_gpu_;
  Camera_gpu* camera_m_gpu = camera_vector_cpu_[index_m]->camera_gpu_;

  int cols = camera_r_cpu->invdepth_map_->image_.cols;
  int rows = camera_r_cpu->invdepth_map_->image_.rows;

  prepareCameraForDtam(index_m);

  dim3 threadsPerBlock( 4 , 4 , NUM_INTERPOLATIONS);
  dim3 numBlocks( rows/4, cols/4 , 1);
  UpdateCostVolume_kernel<<<numBlocks,threadsPerBlock>>>(camera_r_gpu, camera_m_gpu, cost_volume_, camera_data_for_dtam_, invdepth_r_array_, threshold_, occl);
  printCudaError("Kernel updating cost volume");

  double t_e=getTime();
  double delta=t_e-t_s;

  std::cerr << "cost volume computation took: " << delta << " ms " << std::endl;


}

double Mapper::StudyCostVolumeMin(int index_m, int row, int col,bool showbaseline=false){
  double t_s=getTime();


  Camera_cpu* camera_r_cpu = camera_vector_cpu_[index_r_];
  Camera_gpu* camera_r_gpu = camera_r_cpu->camera_gpu_;
  Camera_gpu* camera_m_gpu = camera_vector_cpu_[index_m]->camera_gpu_;

  dim3 threadsPerBlock( 1 , 1 , NUM_INTERPOLATIONS);
  dim3 numBlocks( 1, 1 , 1);
  StudyCostVolumeMin_kernel<<<numBlocks,threadsPerBlock>>>(camera_r_gpu, camera_m_gpu, cost_volume_, camera_data_for_dtam_, invdepth_r_array_,row,col, depth_groundtruth_, a);
  printCudaError("Kernel studying cost volume");

  if(showbaseline){
    Image< cv::Vec3b >* study_baseline = new Image< cv::Vec3b >("Study baseline");
    camera_vector_cpu_[index_m]->image_rgb_gpu_.download(study_baseline->image_);
    study_baseline->show(1500/camera_vector_cpu_[index_m]->resolution_);
    delete study_baseline;

  }



  if(showbaseline){
    Image< cv::Vec3b >* study_ref = new Image< cv::Vec3b >("Study reference");
    camera_vector_cpu_[index_r_]->image_rgb_gpu_.download(study_ref->image_);
    Eigen::Vector2i pxl(col,row);
    cv::Vec3b red(0,0,255);
    study_ref->setPixel(pxl, red);
    study_ref->show(1500/camera_vector_cpu_[index_r_]->resolution_);
  }

  double t_e=getTime();
  double delta=t_e-t_s;
  return delta;
}

void Mapper::ComputeCostVolumeMin(){

  double t_s=getTime();

  Camera_cpu* camera_r_cpu = camera_vector_cpu_[index_r_];
  Camera_gpu* camera_r_gpu = camera_r_cpu->camera_gpu_;
  int cols = camera_r_cpu->invdepth_map_->image_.cols;
  int rows = camera_r_cpu->invdepth_map_->image_.rows;

  dim3 threadsPerBlock( 4 , 4 , NUM_INTERPOLATIONS);
  dim3 numBlocks( rows/4, cols/4 , 1);
  ComputeCostVolumeMin_kernel<<<numBlocks,threadsPerBlock>>>( d, a, cost_volume_, invdepth_r_array_);
  printCudaError("Kernel computing cost volume min");

  double t_e=getTime();
  double delta=t_e-t_s;
  std::cerr << "ComputeCostVolumeMin took: " << delta << " ms " << std::endl;
}



void Mapper::ComputeWeightedGradientSobelImage(cv::cuda::GpuMat* image_in, cv::cuda::GpuMat* image_out){

  int cols = image_in->cols;
  int rows = image_in->rows;

  dim3 threadsPerBlock( 32 , 32 , 1);
  dim3 numBlocks( rows/32, cols/32 , 1);
  ComputeWeightedGradientSobelImage_kernel<<<numBlocks,threadsPerBlock>>>(*image_in, *image_out, weight_matrix_);
  printCudaError("Kernel computing gradient");
}


void Mapper::ComputeWeightedDivergenceSobelImage(cv::cuda::GpuMat* image_in, cv::cuda::GpuMat* image_out){

  int cols = image_in->cols/2;
  int rows = image_in->rows/2;

  dim3 threadsPerBlock( 32 , 32 , 1);
  dim3 numBlocks( rows/32, cols/32 , 1);
  ComputeWeightedDivergenceSobelImage_kernel<<<numBlocks,threadsPerBlock>>>(*image_in, *image_out, weight_matrix_);
  printCudaError("Kernel computing gradient");
}

__global__ void gradDesc_Q_toNormalize_kernel(cv::cuda::PtrStepSz<float> q, cv::cuda::PtrStepSz<float> gradient_d, float eps, float sigma_q, float* vector_to_normalize){
  int row = blockIdx.x * blockDim.x + threadIdx.x;
  int col = blockIdx.y * blockDim.y + threadIdx.y;

  int rows = blockDim.x*gridDim.x;
  int cols = blockDim.y*gridDim.y;

  int index = row+col*rows;
  vector_to_normalize[index]=(q(row,col)+sigma_q*gradient_d(row,col))/(1+sigma_q*eps);
  // vector_to_normalize[index]=(sigma_q*gradient_d(row,col))/(1+sigma_q*eps);
  // vector_to_normalize[index]=1;

}

__global__ void gradDesc_Q_kernel(cv::cuda::PtrStepSz<float> q, cv::cuda::PtrStepSz<float> gradient_d, float eps, float sigma_q ){
  int row = blockIdx.x * blockDim.x + threadIdx.x;
  int col = blockIdx.y * blockDim.y + threadIdx.y;

  int rows = blockDim.x*gridDim.x;
  int cols = blockDim.y*gridDim.y;

  int index = row+col*rows;
  q(row,col)=(q(row,col)+sigma_q*gradient_d(row,col))/(1+sigma_q*eps);

}

__global__ void gradDesc_D_kernel(cv::cuda::PtrStepSz<float> d, cv::cuda::PtrStepSz<float> a, cv::cuda::PtrStepSz<float> gradient_q, float sigma_d, float theta){
  int row = blockIdx.x * blockDim.x + threadIdx.x;
  int col = blockIdx.y * blockDim.y + threadIdx.y;

  int rows = blockDim.x*gridDim.x;
  int cols = blockDim.y*gridDim.y;

  d(row,col)=(d(row,col)+sigma_d*(gradient_q(row,col)+(1.0/theta)*a(row,col)))/(1+(sigma_d/theta));
  // d(row,col)=(d(row,col)+sigma_d*(gradient_q(row,col)+(1.0/theta)*d(row,col)))/(1+(sigma_d/theta));
  // d(row,col)=(d(row,col)+sigma_d*(gradient_q(row,col)))/(1+(sigma_d/theta));
  // d(row,col)=1;

}

__global__ void search_A_kernel(cv::cuda::PtrStepSz<float> d, cv::cuda::PtrStepSz<float> a, cv::cuda::PtrStepSz<int2> cost_volume , float lambda, float theta, float* invdepth_r_array){
  int row = blockIdx.x * blockDim.x + threadIdx.x;
  int col = blockIdx.y * blockDim.y + threadIdx.y;
  int i = blockIdx.z * blockDim.z + threadIdx.z;

  int rows = blockDim.x*gridDim.x;
  int cols = blockDim.y*gridDim.y;

  int col_ = cols*i+col;

  __shared__ float cost_array[4][4][NUM_INTERPOLATIONS];
  __shared__ int indx_array[4][4][NUM_INTERPOLATIONS];

  float a_i = invdepth_r_array[i]/invdepth_r_array[0];

  cost_array[threadIdx.x][threadIdx.y][i]=(1.0/(2*theta))*(d(row,col)-a_i)*(d(row,col)-a_i)+lambda*cost_volume(row,col_).x;
  indx_array[threadIdx.x][threadIdx.y][i]=i;
  __syncthreads();

  // -----------------------------------
  // REDUCTION
  // Iterate of log base 2 the block dimension
	for (int s = 1; s < NUM_INTERPOLATIONS; s *= 2) {
		// Reduce the threads performing work by half previous the previous
		// iteration each cycle
		if (i % (2 * s) == 0) {
      float min_cost = fminf(cost_array[threadIdx.x][threadIdx.y][i + s], cost_array[threadIdx.x][threadIdx.y][i]);
      if (cost_array[threadIdx.x][threadIdx.y][i] > min_cost ){
        indx_array[threadIdx.x][threadIdx.y][i] = indx_array[threadIdx.x][threadIdx.y][i+s];
        cost_array[threadIdx.x][threadIdx.y][i] = min_cost ;
      }
		}
		__syncthreads();
	}

  if (i == indx_array[threadIdx.x][threadIdx.y][0]) {

    // else{
    //   a(row,col)=0.025;
    // }
    a(row,col)=a_i;
    if (cost_volume(row,col_).y==0)
      a(row,col)=1;

	}
  // -----------------------------------
}

// https://github.com/CoffeeBeforeArch/cuda_programming/blob/master/sumReduction/diverged/sumReduction.cu
__global__ void sumReduction_kernel(float *v, float *v_r, int size) {
	// Allocate shared memory
	__shared__ float partial_sum[MAX_THREADS];

	// Calculate thread ID
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	// Load elements into shared memory
  if (tid<size)
  	partial_sum[threadIdx.x] = v[tid];
  else
    partial_sum[threadIdx.x] = 0;

	__syncthreads();

	// Iterate of log base 2 the block dimension
	for (int s = 1; s < blockDim.x; s *= 2) {
		// Reduce the threads performing work by half previous the previous
		// iteration each cycle
		if (threadIdx.x % (2 * s) == 0) {
			partial_sum[threadIdx.x] += partial_sum[threadIdx.x + s];
		}
		__syncthreads();
	}

	// Let the thread 0 for this block write it's result to main memory
	// Result is inexed by this block
	if (threadIdx.x == 0) {
		v_r[blockIdx.x] = partial_sum[0];
	}    // void CostVolumeMin(int num_interpolations);
    // bool get1stDepthWithUV(Camera* camera_r, Camera* camera_m, Eigen::Vector2f& uv_r, Eigen::Vector2f& uv_m, float& depth);

}


__global__ void maxReduction_kernel(float *v, float *v_r, int size) {
	// Allocate shared memory
	__shared__ float partial_max[MAX_THREADS];

	// Calculate thread ID
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	// Load elements into shared memory
  if (tid<size)
  	partial_max[threadIdx.x] = v[tid];
  else
    partial_max[threadIdx.x] = 0;

	__syncthreads();

	// Iterate of log base 2 the block dimension
	for (int s = 1; s < blockDim.x; s *= 2) {
		// Reduce the threads performing work by half previous the previous
		// iteration each cycle
		if (threadIdx.x % (2 * s) == 0) {
			partial_max[threadIdx.x] = fmaxf(abs(partial_max[threadIdx.x + s]),abs(partial_max[threadIdx.x]));
		}
		__syncthreads();
	}

	// Let the thread 0 for this block write it's result to main memory
	// Result is inexed by this block
	if (threadIdx.x == 0) {
		v_r[blockIdx.x] = partial_max[0];
	}
}

__global__ void normalize_Q_kernel(float norm, cv::cuda::PtrStepSz<float> q, float* vector_to_normalize){
  int row = blockIdx.x * blockDim.x + threadIdx.x;
  int col = blockIdx.y * blockDim.y + threadIdx.y;

  int rows = blockDim.x*gridDim.x;
  int cols = blockDim.y*gridDim.y;

  int index = row+col*rows;

  // int scale= 200;
  // float denominator = fmaxf(1,norm/scale);
  // q(row,col)=(vector_to_normalize[index]/denominator);

  q(row,col)=fmaxf(1,vector_to_normalize[index]);

  // q(row,col)=vector_to_normalize[index];

}

__global__ void squareVectorElements_kernel(float *vector){
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  vector[index]=vector[index]*vector[index];

}

__global__ void sqrt_kernel(float* v){
  v[0]=sqrt(v[0]);
}

__global__ void copyArray_kernel(float* original, float* copy){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  copy[index]=original[index];
}

void Mapper::getVectorNorm(float* vector_to_normalize, float* norm, int N){


  int GRID_SIZE = N;
  int N_THREADS = N;

  float* norm_vector_i;
  hipMalloc(&norm_vector_i, sizeof(float)*N);
  printCudaError("hipMalloc in norm computation 1");

  copyArray_kernel<<<N/MAX_THREADS,MAX_THREADS>>>(vector_to_normalize, norm_vector_i);
  printCudaError("Copying array for norm computation");

  squareVectorElements_kernel<<<N/MAX_THREADS,MAX_THREADS>>>(norm_vector_i);
  printCudaError("Squaring terms for computing norm");

  float* norm_vector_o;


  const int TB_SIZE = MAX_THREADS;
  bool init = false;

  while (GRID_SIZE>=TB_SIZE){

    hipMalloc(&norm_vector_o, sizeof(float)*GRID_SIZE);
    printCudaError("hipMalloc in norm computation 2");

    int REST = GRID_SIZE % TB_SIZE;
    GRID_SIZE = GRID_SIZE / TB_SIZE;
    if (REST > 0)
      GRID_SIZE++;
    // N_THREADS = N_THREADS;

    sumReduction_kernel<<<GRID_SIZE, TB_SIZE>>>(norm_vector_i, norm_vector_o, N_THREADS);
    printCudaError("Kernel computing sum reduction for computing norm ");

    N_THREADS = GRID_SIZE;

    if (init)
      hipFree(norm_vector_i);
    norm_vector_i=norm_vector_o;
    init = true;

  }

  hipMalloc(&norm_vector_o, sizeof(float));
  sumReduction_kernel<<<1, TB_SIZE>>>(norm_vector_i, norm_vector_o, N_THREADS);
  printCudaError("Kernel computing sum reduction for computing norm (final)");

  hipDeviceSynchronize();

  sqrt_kernel<<<1, 1>>>(norm_vector_o);
  printCudaError("sqrt for norm computation");

  hipMemcpy(norm, norm_vector_o , sizeof(float), hipMemcpyDeviceToHost);
  printCudaError("Copying result device to host");

  hipFree(norm_vector_i);
  hipFree(norm_vector_o);



}

void Mapper::getVectorMax(float* vector, float* max, int N){


  int GRID_SIZE = N;
  int N_THREADS = N;

  float* max_vector_i;
  hipMalloc(&max_vector_i, sizeof(float)*N);
  printCudaError("hipMalloc in max computation 1");

  copyArray_kernel<<<N/MAX_THREADS,MAX_THREADS>>>(vector, max_vector_i);
  printCudaError("Copying array for max computation");

  float* max_vector_o;


  const int TB_SIZE = MAX_THREADS;
  bool init = false;

  while (GRID_SIZE>=TB_SIZE){

    hipMalloc(&max_vector_o, sizeof(float)*GRID_SIZE);
    printCudaError("hipMalloc in max computation 2");

    int REST = GRID_SIZE % TB_SIZE;
    GRID_SIZE = GRID_SIZE / TB_SIZE;
    if (REST > 0)
      GRID_SIZE++;
    // N_THREADS = N_THREADS;

    maxReduction_kernel<<<GRID_SIZE, TB_SIZE>>>(max_vector_i, max_vector_o, N_THREADS);
    printCudaError("Kernel computing sum reduction for computing max ");

    N_THREADS = GRID_SIZE;

    // if (init)
      hipFree(max_vector_i);
    max_vector_i=max_vector_o;
    init = true;

  }

  hipMalloc(&max_vector_o, sizeof(float));
  maxReduction_kernel<<<1, TB_SIZE>>>(max_vector_i, max_vector_o, N_THREADS);
  printCudaError("Kernel computing sum reduction for computing max (final)");


  hipMemcpy(max, max_vector_o , sizeof(float), hipMemcpyDeviceToHost);
  printCudaError("Copying result device to host");

  hipFree(max_vector_i);
  hipFree(max_vector_o);



}


void Mapper::Image2Vector(cv::cuda::GpuMat* image, float* vector){

  int rows = image->rows;
  int cols = image->cols;

  dim3 threadsPerBlock( 32 , 32 , 1);
  dim3 numBlocks( rows/32, cols/32 , 1);
  Image2Vector_kernel<<<numBlocks,threadsPerBlock>>>( *image, vector);
  printCudaError("Kernel converting image to vector");
}

void Mapper::getImageNorm(cv::cuda::GpuMat* image, float* norm ){

  float* vector;

  int rows = image->rows;
  int cols = image->cols;
  int N = rows*cols;
  hipMalloc(&vector, sizeof(float)*N);

  Mapper::Image2Vector( image, vector);

  Mapper::getVectorNorm(vector, norm, N);

}

void Mapper::gradDesc_Q(cv::cuda::GpuMat* q, cv::cuda::GpuMat* gradient_d ){


    int rows = q->rows;
    int cols = q->cols;

    dim3 threadsPerBlock( 32 , 32 , 1);
    dim3 numBlocks( rows/32, cols/32 , 1);
    gradDesc_Q_kernel<<<numBlocks,threadsPerBlock>>>( *q, *gradient_d, eps_, sigma_q_ );
    printCudaError("Kernel computing next q ");

}

// void Mapper::gradDesc_Q(cv::cuda::GpuMat* q, cv::cuda::GpuMat* gradient_d ){
//
//
//   int rows = q->rows;
//   int cols = q->cols;
//   int N = rows*cols;
//   float* vector_to_normalize;
//   hipMalloc(&vector_to_normalize, sizeof(float)*N);
//
//
//   dim3 threadsPerBlock( 32 , 32 , 1);
//   dim3 numBlocks( rows/32, cols/32 , 1);
//   gradDesc_Q_toNormalize_kernel<<<numBlocks,threadsPerBlock>>>( *q, *gradient_d, eps_, sigma_q_, vector_to_normalize );
//   printCudaError("Kernel computing next q to normalize");
//
//   // float* norm = new float;
//   // Mapper::getVectorNorm(vector_to_normalize, norm, N);
//   // // std::cout << "norm is: " << *norm << std::endl;
//   // normalize_Q_kernel<<<numBlocks,threadsPerBlock>>> (*norm, *q, vector_to_normalize);
//   // printCudaError("Kernel computing sum reduction");
//
//   float* max = new float;
//   Mapper::getVectorMax(vector_to_normalize, max, N);
//   // std::cout << "max is: " << *max << std::endl;
//   normalize_Q_kernel<<<numBlocks,threadsPerBlock>>> (*max, *q, vector_to_normalize);
//   printCudaError("Kernel computing sum reduction");
//
//   hipFree(vector_to_normalize);
//
// }

void Mapper::gradDesc_D(cv::cuda::GpuMat* d, cv::cuda::GpuMat* a, cv::cuda::GpuMat* gradient_q ){


  int rows = d->rows;
  int cols = d->cols;

  dim3 threadsPerBlock( 32 , 32 , 1);
  dim3 numBlocks( rows/32, cols/32 , 1);
  gradDesc_D_kernel<<<numBlocks,threadsPerBlock>>>( *d, *a, *gradient_q, sigma_d_, theta_);
  printCudaError("Kernel computing next d");

}

void Mapper::search_A(cv::cuda::GpuMat* d, cv::cuda::GpuMat* a ){


  int rows = d->rows;
  int cols = d->cols;

  dim3 threadsPerBlock( 4 , 4 , NUM_INTERPOLATIONS);
  dim3 numBlocks( rows/4, cols/4 , 1);
  search_A_kernel<<<numBlocks,threadsPerBlock>>>( *d, *a, cost_volume_, lambda_ , theta_, invdepth_r_array_);
  printCudaError("Kernel computing search on a");

}

void Mapper::ComputeWeights(){

  Camera_cpu* camera_r_cpu = camera_vector_cpu_[index_r_];
  Camera_gpu* camera_r_gpu = camera_r_cpu->camera_gpu_;
  int cols = camera_r_cpu->invdepth_map_->image_.cols;
  int rows = camera_r_cpu->invdepth_map_->image_.rows;
  dim3 threadsPerBlock( 32 , 32 , 1);
  dim3 numBlocks( rows/32, cols/32 , 1);
  ComputeWeights_kernel<<<numBlocks,threadsPerBlock>>>( camera_r_gpu, weight_matrix_, alpha_, beta1_);
  printCudaError("Kernel computing cost volume min");

}

void Mapper::UpdateParametersReg(bool trigger, bool print=false){


  // upgrade theta
  // std::cout << "theta: " << theta_ <<std::endl;
  float beta = trigger ? beta1_ : beta2_;
  float r = trigger ? r1_ : r2_;

  n_++;  // upgrade n

  theta_ = theta_*(1-beta*n_);
  float r_pow_n=pow(r,n_);
  sigma_q_=sigma_q0_/r_pow_n;
  sigma_d_=sigma_d0_*r_pow_n;

  if(print){
    std::cout << "theta_: " << theta_ << std::endl;
    std::cout << "sigma_q_: " << sigma_q_ << std::endl;
    std::cout << "sigma_d_: " << sigma_d_ << std::endl;
    std::cout << "n_: " << n_ << std::endl;
    std::cout << "beta: " << beta << std::endl;
    std::cout << "r: " << r << std::endl;
  }

}


bool Mapper::Regularize(){

  if(theta_>theta_end_){
    double t_s=getTime();


    int resolution=camera_vector_cpu_[index_r_]->resolution_;

    // cv::cuda::GpuMat a0 = camera_vector_cpu_[index_r_]->invinvdepth_map_gpu_.clone();
    // cv::Mat_< float > a_0;
    // (a0).download(a_0);
    // cv::Mat_< float > resized_image_a_0;
    // cv::resize(a_0, resized_image_a_0, cv::Size(), 800/resolution, 800/resolution, cv::INTER_NEAREST );
    // a_0.convertTo(a_0, CV_32FC1, 255.0);
    // cv::imwrite("/home/manu/Desktop/dtam_thesis/matlab_scripts/data2.png", a_0);
    //
    // cv::Mat_< float > d_0;
    // (d).download(d_0);
    // cv::Mat_< float > resized_image_d_0;
    // cv::resize(d_0, resized_image_d_0, cv::Size(), 800/resolution, 800/resolution, cv::INTER_NEAREST );
    // cv::imshow("d_0", resized_image_d_0);

    // float* norm_d0 = new float;
    // Mapper::getImageNorm(&d, norm_d0);
    // std::cout << "\nd norm 0 is: " << *norm_d0 << std::endl;


    Mapper::ComputeWeightedGradientSobelImage( &d, &gradient_d ); // compute gradient of d (n)

    Mapper::gradDesc_Q( &q, &gradient_d);  // compute q (n+1)

    Mapper::ComputeWeightedDivergenceSobelImage( &q, &gradient_q ); // compute gradient of q (n+1)

    Mapper::gradDesc_D( &d, &a, &gradient_q);  // compute d (n+1)


    Mapper::search_A( &d, &a );  // compute a (n+1)

    // Mapper::UpdateParametersReg( theta_>theta_switch_ ,false);
    // Mapper::UpdateParametersReg( frames_computed_<30 ,false);
    Mapper::UpdateParametersReg( true ,false);


    // float* norm_d = new float;
    // Mapper::getImageNorm(&d, norm_d);
    // std::cout << "\nd norm is: " << *norm_d << std::endl;
    //
    // float* norm_q = new float;
    // Mapper::getImageNorm(&q, norm_q);
    // std::cout << "q norm is: " << *norm_q << std::endl;
    //
    // float* norm_sobel_d = new float;
    // Mapper::getImageNorm(gradient_d, norm_sobel_d);
    // std::cout << "sobel d norm is: " << *norm_sobel_d << std::endl;
    //
    // std::cout << "theta is: " << theta_ << std::endl;



    // camera_vector_cpu_[index_r_]->invdepth_map_gpu_= a;


    double t_e=getTime();
    double delta=t_e-t_s;
    std::cerr << "Regularize took: " << delta << " ms " << std::endl;

    return true;
  }
  return false;
}

void Mapper::UpdateDepthmap(){

  double t_s=getTime();

  Camera_cpu* camera_cpu = camera_vector_cpu_[index_r_];
  Camera_gpu* camera_gpu = camera_cpu->camera_gpu_;
  int cols = camera_cpu->invdepth_map_->image_.cols;
  int rows = camera_cpu->invdepth_map_->image_.rows;
  float depth1_r=camera_cpu->min_depth_;
  float depth2_r=camera_cpu->max_depth_;

  dim3 threadsPerBlock( 32 , 32 , 1);
  dim3 numBlocks( rows/32, cols/32 , 1);
  UpdateDepthmap_kernel<<<numBlocks,threadsPerBlock>>>( camera_gpu, cost_volume_, weight_matrix_, a, d,  gradient_q, switch_idx_, switch_depth_, depth1_r, depth2_r);
  printCudaError("State Update State");

  double t_e=getTime();
  double delta=t_e-t_s;
  std::cerr << "State Update: " << delta << " ms " << std::endl;


}

void Mapper::PopulateState(){

  double t_s=getTime();

  Camera_cpu* camera_cpu = camera_vector_cpu_[index_r_];
  Camera_gpu* camera_gpu = camera_cpu->camera_gpu_;
  int cols = camera_cpu->invdepth_map_->image_.cols;
  int rows = camera_cpu->invdepth_map_->image_.rows;
  int n_pixels = cols*rows;

  dim3 threadsPerBlock( 32 , 32 , 1);
  dim3 numBlocks( rows/32, cols/32 , 1);
  PopulateState_kernel<<<numBlocks,threadsPerBlock>>>( camera_gpu );
  printCudaError("Populate Update Kernel");

  camera_cpu->cp_array_ = new Cp_gpu[n_pixels];
  hipMemcpy(camera_cpu->cp_array_, camera_cpu->cp_array_gpu_ , sizeof(Cp_gpu)*n_pixels, hipMemcpyDeviceToHost);
  printCudaError("Copying cp_array device to host");

  hipDeviceSynchronize();

  double t_e=getTime();
  double delta=t_e-t_s;
  std::cerr << "Populate State: " << delta << " ms " << std::endl;

}

void Mapper::StateFromGt(){
  Camera_cpu* camera = camera_vector_cpu_[index_r_];
  depth_groundtruth_.download(camera->invdepth_map_->image_);
  camera->getCamera_gpu();
}
void Mapper::depthSampling(Environment_gpu* environment){
  int rows = environment->resolution_/environment->aspect_;
  int cols = environment->resolution_;
  float depth1_r=environment->min_depth_;
  float depth2_r=environment->max_depth_;
  float* invdepth_r_array_h = new float[NUM_INTERPOLATIONS];

  switch_idx_=40;
  switch_depth_=5;
  for (int i=0; i<switch_idx_; i++){
    float ratio_depth_r = (float)(i)/((float)switch_idx_);
    float depth_r = depth1_r+ratio_depth_r*(switch_depth_-depth1_r);
    invdepth_r_array_h[i]=1.0/depth_r;
    std::cout << "depth: " << depth_r  << ", idx: " << i << std::endl;
  }
  for (int i=switch_idx_; i<NUM_INTERPOLATIONS; i++){
    float ratio_depth_r = (float)(i-switch_idx_)/((float)NUM_INTERPOLATIONS-switch_idx_-1);
    float invdepth_r = (1.0/switch_depth_)+ratio_depth_r*((1.0/depth2_r)-(1.0/switch_depth_));
    float depth_r = 1.0/invdepth_r;
    invdepth_r_array_h[i]=1.0/depth_r;
    std::cout << "depth: " << depth_r  << ", idx: " << i << std::endl;
  }

  hipError_t err ;

  hipMalloc(&invdepth_r_array_, sizeof(float)*NUM_INTERPOLATIONS);
  err = hipGetLastError();
  if (err != hipSuccess)
      printf("hipMalloc (dtam constr) Error: %s\n", hipGetErrorString(err));

  hipMemcpy(invdepth_r_array_, invdepth_r_array_h, sizeof(float)*NUM_INTERPOLATIONS, hipMemcpyHostToDevice);
  err = hipGetLastError();
  if (err != hipSuccess)
      printf("hipMemcpy (dtam constr) Error: %s\n", hipGetErrorString(err));


  delete (invdepth_r_array_h);

}
