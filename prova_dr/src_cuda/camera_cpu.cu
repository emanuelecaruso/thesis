#include "camera_cpu.cuh"
#include <thread>
#include <vector>
#include <mutex>

using namespace std;
using namespace pr;



void Camera_cpu::gpuFree(){
  image_rgb_gpu_.release();
  invdepth_map_gpu_.release();
}


void Camera_cpu::getCamera_gpu(){
  hipError_t err ;

  image_rgb_gpu_.upload(image_rgb_->image_);
  invdepth_map_gpu_.upload(invdepth_map_->image_);

  int n_pixels=resolution_*(resolution_/aspect_);

  hipMalloc(&cp_array_gpu_, sizeof(Cp_gpu)*n_pixels);
  err = hipGetLastError();
  if (err != hipSuccess)
      printf("hipMalloc cps Error: %s\n", hipGetErrorString(err));

  Camera_gpu* camera_gpu_h = new Camera_gpu(name_, lens_, aspect_, width_, resolution_,
     max_depth_,min_depth_, K_, Kinv_, *frame_camera_wrt_world_, *frame_world_wrt_camera_,
     *frame_camera_wrt_world_gt_, *frame_world_wrt_camera_gt_, invdepth_map_gpu_,
     image_rgb_gpu_, cp_array_gpu_);


  hipMalloc((void**)&camera_gpu_, sizeof(Camera_gpu));
  err = hipGetLastError();
  if (err != hipSuccess)
      printf("hipMalloc %s%s",name_," Error: %s\n", hipGetErrorString(err));

  hipMemcpy(camera_gpu_, camera_gpu_h, sizeof(Camera_gpu), hipMemcpyHostToDevice);
  err = hipGetLastError();
  if (err != hipSuccess)
      printf("hipMemcpy %s%s",name_," Error: %s\n", hipGetErrorString(err));

  delete camera_gpu_h;
}

void Camera_cpu::cloneCameraImages(Camera* camera){
  invdepth_map_ = camera->invdepth_map_;
  image_rgb_ = camera->image_rgb_;

}

void Camera_cpu::showInvdepthmap(int scale){
  Image<float>* invdepthmap=new Image< float >("invdepth_"+name_);
  invdepth_map_gpu_.download(invdepthmap->image_);
  invdepthmap->show(scale/resolution_);
}

void Camera_cpu::setGroundtruthPose(){
  hipFree(camera_gpu_);

  *frame_world_wrt_camera_=*frame_world_wrt_camera_gt_;
  *frame_camera_wrt_world_=*frame_camera_wrt_world_gt_;

  Camera_cpu::getCamera_gpu();

}
